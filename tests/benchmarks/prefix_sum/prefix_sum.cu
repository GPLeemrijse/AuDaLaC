#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 256
#define ATOMIC(T) cuda::atomic<T, cuda::thread_scope_device>
#define STORE(A, V) A.store(V, cuda::memory_order_relaxed)
#define LOAD(A) A.load(cuda::memory_order_relaxed)

#define WLOAD(T, A) *((T*)&A)
#define WSTORE(T, A, V) *((T*)&A) = V

#define ListElem_MASK (((uint16_t)1) << 0)
#define STEP_PARITY(STRUCT) ((bool)(struct_step_parity & STRUCT ## _MASK))
#define TOGGLE_STEP_PARITY(STRUCT) {struct_step_parity ^= STRUCT ## _MASK;}


#include "ADL.h"
#include "Struct.h"
#include "init_file.h"
#include <hip/hip_cooperative_groups.h>
#include <cuda/atomic>
#include <stdio.h>
#include <tuple>
#include <vector>


class ListElem : public Struct {
public:
	ListElem (void) : Struct() {}
	
	ATOMIC(IntType)* val;
	ATOMIC(RefType)* prev;
	ATOMIC(RefType)* auxprev;
	ATOMIC(IntType)* auxval;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "ListElem");
		assert (info->parameter_types.size() == 4);
		assert (info->parameter_types[0] == ADL::Int);
		assert (info->parameter_types[1] == ADL::Ref);
		assert (info->parameter_types[2] == ADL::Ref);
		assert (info->parameter_types[3] == ADL::Int);
	};

	void** get_parameters(void) {
		return (void**)&val;
	}

	size_t child_size(void) {
		return sizeof(ListElem);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[4] = {
			sizeof(ATOMIC(IntType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(IntType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(IntType _val,
									   RefType _prev,
									   RefType _auxprev,
									   IntType _auxval,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(val[slot], _val);
		STORE(prev[slot], _prev);
		STORE(auxprev[slot], _auxprev);
		STORE(auxval[slot], _auxval);
		*stable = false;
		return slot;
	}
};

using namespace cooperative_groups;

ListElem host_ListElem = ListElem();

ListElem* host_ListElem_ptr = &host_ListElem;

__device__ ListElem* __restrict__ listelem;


#define FP_DEPTH 1
/* Transform an iter_idx into the fp_stack index
   associated with that operation.
*/
#define FP_SET(X) (X)
#define FP_RESET(X) ((X) + 1 >= 3 ? (X) + 1 - 3 : (X) + 1)
#define FP_READ(X) ((X) + 2 >= 3 ? (X) + 2 - 3 : (X) + 2)

__device__ cuda::atomic<bool, cuda::thread_scope_device> fp_stack[FP_DEPTH][3];

__device__ void clear_stack(int lvl, uint8_t* iter_idx) {
	/*	Clears the stack on the FP_SET side.
		The FP_RESET and FP_READ sides should remain the same.
	*/
	while(lvl >= 0){
		fp_stack[lvl][FP_SET(iter_idx[lvl])].store(false, cuda::memory_order_relaxed);
		lvl--;
	}
}

typedef void(*step_func)(RefType, bool*);
template <step_func Step>
__device__ void executeStep(inst_size nrof_instances, grid_group grid, thread_block block, bool* stable){
	for(RefType self = grid.thread_rank(); self < nrof_instances; self += grid.size()){

		Step(self, stable);
    }
}

__host__ std::tuple<dim3, dim3> get_launch_dims(inst_size max_nrof_executing_instances, const void* kernel){
  int numBlocksPerSm = 0;
  int tpb = THREADS_PER_BLOCK;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, kernel, tpb, 0);
  
  int max_blocks = deviceProp.multiProcessorCount*numBlocksPerSm;
  int wanted_blocks = (max_nrof_executing_instances + tpb - 1)/tpb;
  int used_blocks = min(max_blocks, wanted_blocks);

  fprintf(stderr, "Launching %u/%u blocks of %u threads = %u threads.\nResulting in max %u instances per thread.\n", used_blocks, max_blocks, tpb, used_blocks * tpb, (max_nrof_executing_instances + (used_blocks * tpb) - 1) / (used_blocks * tpb));

  dim3 dimBlock(tpb, 1, 1);
  dim3 dimGrid(used_blocks, 1, 1);
  return std::make_tuple(dimGrid, dimBlock);
}

template<typename T>
__device__ void SetParam(const RefType owner, ATOMIC(T) * const params, const T new_val, bool* stable) {
	if (owner != 0){
		T old_val = LOAD(params[owner]);
		if (old_val != new_val){
			STORE(params[owner], new_val);
			*stable = false;
		}
	}
}

template<typename T>
__device__ void WSetParam(const RefType owner, ATOMIC(T) * const params, const T new_val, bool* stable) {
    if (owner != 0){
        T old_val = WLOAD(T, params[owner]);
        if (old_val != new_val){
            WSTORE(T, params[owner], new_val);
            *stable = false;
        }
    }
}

__device__ void ListElem_print(const RefType self,
							   bool* stable){
	if (self != 0) {
		printf("ListElem(%u): val=%d, prev=%u, auxprev=%u, auxval=%d\n", self, LOAD(listelem->val[self]), LOAD(listelem->prev[self]), LOAD(listelem->auxprev[self]), LOAD(listelem->auxval[self]));
	}
}

__device__ void ListElem_prefixOne(const RefType self,
								   bool* stable){
	
	// auxval := prev.val;
	WSetParam<IntType>(self, listelem->auxval, WLOAD(IntType, listelem->val[WLOAD(RefType, listelem->prev[self])]), stable);
	// auxprev := prev.prev;
	WSetParam<RefType>(self, listelem->auxprev, WLOAD(RefType, listelem->prev[WLOAD(RefType, listelem->prev[self])]), stable);
}

__device__ void ListElem_prefixTwo(const RefType self,
								   bool* stable){
	
	// val := val + auxval;
	WSetParam<IntType>(self, listelem->val, (WLOAD(IntType, listelem->val[self]) + WLOAD(IntType, listelem->auxval[self])), stable);
	// prev := auxprev;
	WSetParam<RefType>(self, listelem->prev, WLOAD(RefType, listelem->auxprev[self]), stable);
}

__device__ void ListElem_print_sol(const RefType self,
								   bool* stable){
	
	if ((self != 0)) {
		IntType ld_val = WLOAD(IntType, listelem->val[self]);
		printf("(%u, %d)\n", self, ld_val);
	}
}


__global__ void schedule_kernel(){
	const grid_group grid = this_grid();
	const thread_block block = this_thread_block();
	uint16_t struct_step_parity = 0; // bitmask
	bool stable = true; // Only used to compile steps outside fixpoints
	uint8_t iter_idx[FP_DEPTH] = {0}; // Denotes which fp_stack index ([0, 2]) is currently being set.

	do{
		bool stable = true;
		if (grid.thread_rank() == 0){
			/* Resets the next fp_stack index in advance. */
			fp_stack[0][FP_RESET(iter_idx[0])].store(true, cuda::memory_order_relaxed);
		}


		TOGGLE_STEP_PARITY(ListElem);
		executeStep<ListElem_prefixOne>(listelem->nrof_instances2(STEP_PARITY(ListElem)), grid, block, &stable);
		listelem->update_counters(!STEP_PARITY(ListElem));

		grid.sync();

		TOGGLE_STEP_PARITY(ListElem);
		executeStep<ListElem_prefixTwo>(listelem->nrof_instances2(STEP_PARITY(ListElem)), grid, block, &stable);
		listelem->update_counters(!STEP_PARITY(ListElem));
		if(!stable){
			clear_stack(0, iter_idx);
		}
		/* The next index to set is the one that has been reset. */
		iter_idx[0] = FP_RESET(iter_idx[0]);
		grid.sync();
	} while(!fp_stack[0][FP_READ(iter_idx[0])].load(cuda::memory_order_relaxed));


	TOGGLE_STEP_PARITY(ListElem);
	executeStep<ListElem_print_sol>(listelem->nrof_instances2(STEP_PARITY(ListElem)), grid, block, &stable);
	listelem->update_counters(!STEP_PARITY(ListElem));
}


int main(int argc, char **argv) {
	if (argc != 2) {
		printf("Supply a .init file.\n");
		exit(1);
	}

	std::vector<InitFile::StructInfo> structs = InitFile::parse(argv[1]);
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 1048576);
	CHECK(hipHostRegister(&host_ListElem, sizeof(ListElem), hipHostRegisterDefault));

	host_ListElem.initialise(&structs[0], structs[0].nrof_instances + 1);

	inst_size max_nrof_executing_instances = structs[0].nrof_instances + 1;
	CHECK(hipDeviceSynchronize());

	ListElem * const loc_listelem = (ListElem*)host_ListElem.to_device();

	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(listelem), &loc_listelem, sizeof(ListElem * const)));

	cuda::atomic<bool, cuda::thread_scope_device>* fp_stack_address;
	CHECK(hipGetSymbolAddress((void **)&fp_stack_address, fp_stack));
	CHECK(hipMemset((void*)fp_stack_address, 1, FP_DEPTH * 3 * sizeof(cuda::atomic<bool, cuda::thread_scope_device>)));
	void* schedule_kernel_args[] = {};
	auto dims = get_launch_dims(max_nrof_executing_instances, (void*)schedule_kernel);


	CHECK(
		hipLaunchCooperativeKernel(
			(void*)schedule_kernel,
			std::get<0>(dims),
			std::get<1>(dims),
			schedule_kernel_args
		)
	);


	CHECK(hipDeviceSynchronize());
}
