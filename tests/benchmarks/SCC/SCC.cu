#include "hip/hip_runtime.h"
#define I_PER_THREAD 16
#define THREADS_PER_BLOCK 256
#define ATOMIC(T) cuda::atomic<T, cuda::thread_scope_device>
#define STORE(A, B) A.store(B, cuda::memory_order_relaxed)
#define LOAD(A) A.load(cuda::memory_order_relaxed)
#define FP_DEPTH 2
#define NodeSet_MASK (1ULL << 0)
#define Node_MASK (1ULL << 1)
#define Edge_MASK (1ULL << 2)
#define STEP_PARITY(STRUCT) ((bool)(struct_step_parity & STRUCT ## _MASK))
#define TOGGLE_STEP_PARITY(STRUCT) {struct_step_parity ^= STRUCT ## _MASK;}


#include "ADL.h"
#include "Struct.h"
#include "init_file.h"
#include <hip/hip_cooperative_groups.h>
#include <cuda/atomic>
#include <stdio.h>
#include <vector>


class NodeSet : public Struct {
public:
	NodeSet (void) : Struct() {}
	
	ATOMIC(RefType)* pivot_f_b;
	ATOMIC(RefType)* pivot_f_nb;
	ATOMIC(RefType)* pivot_nf_b;
	ATOMIC(RefType)* pivot_nf_nb;
	ATOMIC(BoolType)* scc;
	ATOMIC(RefType)* f_and_b;
	ATOMIC(RefType)* not_f_and_b;
	ATOMIC(RefType)* f_and_not_b;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "NodeSet");
		assert (info->parameter_types.size() == 8);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Ref);
		assert (info->parameter_types[2] == ADL::Ref);
		assert (info->parameter_types[3] == ADL::Ref);
		assert (info->parameter_types[4] == ADL::Bool);
		assert (info->parameter_types[5] == ADL::Ref);
		assert (info->parameter_types[6] == ADL::Ref);
		assert (info->parameter_types[7] == ADL::Ref);
	};

	void** get_parameters(void) {
		return (void**)&pivot_f_b;
	}

	size_t child_size(void) {
		return sizeof(NodeSet);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[8] = {
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(BoolType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(RefType _pivot_f_b,
									   RefType _pivot_f_nb,
									   RefType _pivot_nf_b,
									   RefType _pivot_nf_nb,
									   BoolType _scc,
									   RefType _f_and_b,
									   RefType _not_f_and_b,
									   RefType _f_and_not_b,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(pivot_f_b[slot], _pivot_f_b);
		STORE(pivot_f_nb[slot], _pivot_f_nb);
		STORE(pivot_nf_b[slot], _pivot_nf_b);
		STORE(pivot_nf_nb[slot], _pivot_nf_nb);
		STORE(scc[slot], _scc);
		STORE(f_and_b[slot], _f_and_b);
		STORE(not_f_and_b[slot], _not_f_and_b);
		STORE(f_and_not_b[slot], _f_and_not_b);
		*stable = false;
		return slot;
	}
};

class Node : public Struct {
public:
	Node (void) : Struct() {}
	
	ATOMIC(RefType)* set;
	ATOMIC(BoolType)* fwd;
	ATOMIC(BoolType)* bwd;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "Node");
		assert (info->parameter_types.size() == 3);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Bool);
		assert (info->parameter_types[2] == ADL::Bool);
	};

	void** get_parameters(void) {
		return (void**)&set;
	}

	size_t child_size(void) {
		return sizeof(Node);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[3] = {
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(BoolType)),
			sizeof(ATOMIC(BoolType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(RefType _set,
									   BoolType _fwd,
									   BoolType _bwd,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(set[slot], _set);
		STORE(fwd[slot], _fwd);
		STORE(bwd[slot], _bwd);
		*stable = false;
		return slot;
	}
};

class Edge : public Struct {
public:
	Edge (void) : Struct() {}
	
	ATOMIC(RefType)* s;
	ATOMIC(RefType)* t;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "Edge");
		assert (info->parameter_types.size() == 2);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Ref);
	};

	void** get_parameters(void) {
		return (void**)&s;
	}

	size_t child_size(void) {
		return sizeof(Edge);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[2] = {
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(RefType _s,
									   RefType _t,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(s[slot], _s);
		STORE(t[slot], _t);
		*stable = false;
		return slot;
	}
};

using namespace cooperative_groups;

Edge host_Edge = Edge();
Node host_Node = Node();
NodeSet host_NodeSet = NodeSet();

Edge* host_Edge_ptr = &host_Edge;
Node* host_Node_ptr = &host_Node;
NodeSet* host_NodeSet_ptr = &host_NodeSet;

__device__ Edge* __restrict__ edge;
__device__ Node* __restrict__ node;
__device__ NodeSet* __restrict__ nodeset;

/* Transform an iter_idx into the fp_stack index
   associated with that operation.
*/
#define FP_SET(X) (X)
#define FP_RESET(X) ((X) + 1 >= 3 ? (X) + 1 - 3 : (X) + 1)
#define FP_READ(X) ((X) + 2 >= 3 ? (X) + 2 - 3 : (X) + 2)

__device__ cuda::atomic<bool, cuda::thread_scope_device> fp_stack[FP_DEPTH][3];

__device__ __inline__ void clear_stack(int lvl, uint8_t* iter_idx) {
	/*	Clears the stack on the FP_SET side.
		The FP_RESET and FP_READ sides should remain the same.
	*/
	while(lvl >= 0){
		fp_stack[lvl][FP_SET(iter_idx[lvl])].store(false, cuda::memory_order_relaxed);
		lvl--;
	}
}

typedef void(*step_func)(RefType, bool*);
template <step_func Step>
__device__ void executeStep(inst_size nrof_instances, grid_group grid, thread_block block, bool* stable){
	for(int i = 0; i < I_PER_THREAD; i++){
		const RefType self = block.size() * (i + grid.block_rank() * I_PER_THREAD) + block.thread_rank();
		if (self >= nrof_instances) break;

		Step(self, stable);
	}
}
template<typename T>
__device__ void SetParam(const RefType owner, ATOMIC(T) * const params, const T new_val, bool* stable) {
    if (owner != 0){
    	T old_val = LOAD(params[owner]);
    	if (old_val != new_val){
    		STORE(params[owner], new_val);
    		*stable = false;
    	}
    }
}

__device__ __inline__ void print_NodeSet(const RefType self,
										 bool* stable){
	if (self != 0) {
		printf("NodeSet(%u): pivot_f_b=%u, pivot_f_nb=%u, pivot_nf_b=%u, pivot_nf_nb=%u, scc=%u, f_and_b=%u, not_f_and_b=%u, f_and_not_b=%u\n", self, LOAD(nodeset->pivot_f_b[self]), LOAD(nodeset->pivot_f_nb[self]), LOAD(nodeset->pivot_nf_b[self]), LOAD(nodeset->pivot_nf_nb[self]), LOAD(nodeset->scc[self]), LOAD(nodeset->f_and_b[self]), LOAD(nodeset->not_f_and_b[self]), LOAD(nodeset->f_and_not_b[self]));
	}
}

__device__ void NodeSet_allocate_sets(const RefType self,
									  bool* stable){
	
	if ((LOAD(nodeset->pivot_f_b[self]) != (RefType)0)) {
		if ((LOAD(nodeset->pivot_nf_nb[self]) == (RefType)0)) {
			// f_and_b := this;
			SetParam(self, nodeset->f_and_b, self, stable);
			// scc := true;
			SetParam(self, nodeset->scc, true, stable);
			// pivot_f_b := null;
			SetParam(self, nodeset->pivot_f_b, (RefType)0, stable);
		}
		if ((LOAD(nodeset->pivot_nf_nb[self]) != (RefType)0)) {
			// f_and_b := NodeSet(null, null, null, null, true, null, null, null);
			SetParam(self, nodeset->f_and_b, nodeset->create_instance((RefType)0, (RefType)0, (RefType)0, (RefType)0, true, (RefType)0, (RefType)0, (RefType)0, stable), stable);
			// pivot_f_b := null;
			SetParam(self, nodeset->pivot_f_b, (RefType)0, stable);
		}
	}
	if ((LOAD(nodeset->pivot_f_nb[self]) != (RefType)0)) {
		// f_and_not_b := NodeSet(null, pivot_f_nb, null, null, false, null, null, null);
		SetParam(self, nodeset->f_and_not_b, nodeset->create_instance((RefType)0, LOAD(nodeset->pivot_f_nb[self]), (RefType)0, (RefType)0, false, (RefType)0, (RefType)0, (RefType)0, stable), stable);
		// pivot_f_nb := null;
		SetParam(self, nodeset->pivot_f_nb, (RefType)0, stable);
	}
	if ((LOAD(nodeset->pivot_nf_b[self]) != (RefType)0)) {
		// not_f_and_b := NodeSet(null, null, pivot_nf_b, null, false, null, null, null);
		SetParam(self, nodeset->not_f_and_b, nodeset->create_instance((RefType)0, (RefType)0, LOAD(nodeset->pivot_nf_b[self]), (RefType)0, false, (RefType)0, (RefType)0, (RefType)0, stable), stable);
		// pivot_nf_b := null;
		SetParam(self, nodeset->pivot_nf_b, (RefType)0, stable);
	}
}

__device__ void NodeSet_initialise_pivot_fwd_bwd(const RefType self,
												 bool* stable){
	
	if ((!LOAD(nodeset->scc[self]))) {
		// pivot_f_b.fwd := true;
		SetParam(LOAD(nodeset->pivot_f_b[self]), node->fwd, true, stable);
		// pivot_f_b.bwd := true;
		SetParam(LOAD(nodeset->pivot_f_b[self]), node->bwd, true, stable);
		// pivot_f_b := null;
		SetParam(self, nodeset->pivot_f_b, (RefType)0, stable);
		// pivot_f_nb.fwd := true;
		SetParam(LOAD(nodeset->pivot_f_nb[self]), node->fwd, true, stable);
		// pivot_f_nb.bwd := true;
		SetParam(LOAD(nodeset->pivot_f_nb[self]), node->bwd, true, stable);
		// pivot_f_nb := null;
		SetParam(self, nodeset->pivot_f_nb, (RefType)0, stable);
		// pivot_nf_b.fwd := true;
		SetParam(LOAD(nodeset->pivot_nf_b[self]), node->fwd, true, stable);
		// pivot_nf_b.bwd := true;
		SetParam(LOAD(nodeset->pivot_nf_b[self]), node->bwd, true, stable);
		// pivot_nf_b := null;
		SetParam(self, nodeset->pivot_nf_b, (RefType)0, stable);
		// pivot_nf_nb.fwd := true;
		SetParam(LOAD(nodeset->pivot_nf_nb[self]), node->fwd, true, stable);
		// pivot_nf_nb.bwd := true;
		SetParam(LOAD(nodeset->pivot_nf_nb[self]), node->bwd, true, stable);
		// pivot_nf_nb := null;
		SetParam(self, nodeset->pivot_nf_nb, (RefType)0, stable);
	}
}

__device__ __inline__ void print_Node(const RefType self,
									  bool* stable){
	if (self != 0) {
		printf("Node(%u): set=%u, fwd=%u, bwd=%u\n", self, LOAD(node->set[self]), LOAD(node->fwd[self]), LOAD(node->bwd[self]));
	}
}

__device__ void Node_pivots_nominate(const RefType self,
									 bool* stable){
	
	if ((!LOAD(nodeset->scc[LOAD(node->set[self])]))) {
		BoolType f = LOAD(node->fwd[self]);
		BoolType b = LOAD(node->bwd[self]);
		if ((f && b)) {
			// set.pivot_f_b := this;
			SetParam(LOAD(node->set[self]), nodeset->pivot_f_b, self, stable);
		}
		if ((f && (!b))) {
			// set.pivot_f_nb := this;
			SetParam(LOAD(node->set[self]), nodeset->pivot_f_nb, self, stable);
		}
		if (((!f) && b)) {
			// set.pivot_nf_b := this;
			SetParam(LOAD(node->set[self]), nodeset->pivot_nf_b, self, stable);
		}
		if (((!f) && (!b))) {
			// set.pivot_nf_nb := this;
			SetParam(LOAD(node->set[self]), nodeset->pivot_nf_nb, self, stable);
		}
	}
}

__device__ void Node_divide_into_sets_reset_fwd_bwd(const RefType self,
													bool* stable){
	
	BoolType f = LOAD(node->fwd[self]);
	BoolType b = LOAD(node->bwd[self]);
	if ((f && b)) {
		// set := set.f_and_b;
		SetParam(self, node->set, LOAD(nodeset->f_and_b[LOAD(node->set[self])]), stable);
	}
	if (((!f) && b)) {
		// set := set.not_f_and_b;
		SetParam(self, node->set, LOAD(nodeset->not_f_and_b[LOAD(node->set[self])]), stable);
	}
	if ((f && (!b))) {
		// set := set.f_and_not_b;
		SetParam(self, node->set, LOAD(nodeset->f_and_not_b[LOAD(node->set[self])]), stable);
	}
	// fwd := false;
	SetParam(self, node->fwd, false, stable);
	// bwd := false;
	SetParam(self, node->bwd, false, stable);
}

__device__ __inline__ void print_Edge(const RefType self,
									  bool* stable){
	if (self != 0) {
		printf("Edge(%u): s=%u, t=%u\n", self, LOAD(edge->s[self]), LOAD(edge->t[self]));
	}
}

__device__ void Edge_compute_fwd_bwd(const RefType self,
									 bool* stable){
	
	if ((LOAD(node->set[LOAD(edge->t[self])]) == LOAD(node->set[LOAD(edge->s[self])]))) {
		if (LOAD(node->fwd[LOAD(edge->s[self])])) {
			// t.fwd := true;
			SetParam(LOAD(edge->t[self]), node->fwd, true, stable);
		}
		if (LOAD(node->bwd[LOAD(edge->t[self])])) {
			// s.bwd := true;
			SetParam(LOAD(edge->s[self]), node->bwd, true, stable);
		}
	}
}


__global__ void schedule_kernel(){
	const grid_group grid = this_grid();
	const thread_block block = this_thread_block();
	const bool is_thread0 = grid.thread_rank() == 0;
	inst_size nrof_instances;
	uint64_t struct_step_parity = 0; // bitmask
	bool stable = true; // Only used to compile steps outside fixpoints
	uint8_t iter_idx[FP_DEPTH] = {0}; // Denotes which fp_stack index ([0, 2]) is currently being set.

	do{
		bool stable = true;
		if (is_thread0){
			/* Resets the next fp_stack index in advance. */
			fp_stack[0][FP_RESET(iter_idx[0])].store(true, cuda::memory_order_relaxed);
		}


		TOGGLE_STEP_PARITY(Node);
		nrof_instances = node->nrof_instances2(STEP_PARITY(Node));
		executeStep<Node_pivots_nominate>(nrof_instances, grid, block, &stable);
		node->update_counters(!STEP_PARITY(Node));

		grid.sync();

		TOGGLE_STEP_PARITY(NodeSet);
		nrof_instances = nodeset->nrof_instances2(STEP_PARITY(NodeSet));
		executeStep<NodeSet_allocate_sets>(nrof_instances, grid, block, &stable);
		nodeset->update_counters(!STEP_PARITY(NodeSet));

		grid.sync();

		TOGGLE_STEP_PARITY(Node);
		nrof_instances = node->nrof_instances2(STEP_PARITY(Node));
		executeStep<Node_divide_into_sets_reset_fwd_bwd>(nrof_instances, grid, block, &stable);
		node->update_counters(!STEP_PARITY(Node));

		grid.sync();

		TOGGLE_STEP_PARITY(NodeSet);
		nrof_instances = nodeset->nrof_instances2(STEP_PARITY(NodeSet));
		executeStep<NodeSet_initialise_pivot_fwd_bwd>(nrof_instances, grid, block, &stable);
		nodeset->update_counters(!STEP_PARITY(NodeSet));

		grid.sync();

		do{
			bool stable = true;
			if (is_thread0){
				/* Resets the next fp_stack index in advance. */
				fp_stack[1][FP_RESET(iter_idx[1])].store(true, cuda::memory_order_relaxed);
			}


			TOGGLE_STEP_PARITY(Edge);
			nrof_instances = edge->nrof_instances2(STEP_PARITY(Edge));
			executeStep<Edge_compute_fwd_bwd>(nrof_instances, grid, block, &stable);
			edge->update_counters(!STEP_PARITY(Edge));
			if(!stable){
				clear_stack(1, iter_idx);
			}
			/* The next index to set is the one that has been reset. */
			iter_idx[1] = FP_RESET(iter_idx[1]);
			grid.sync();
		} while(!fp_stack[1][FP_READ(iter_idx[1])].load(cuda::memory_order_relaxed));

		if(!stable){
			clear_stack(0, iter_idx);
		}
		/* The next index to set is the one that has been reset. */
		iter_idx[0] = FP_RESET(iter_idx[0]);
		grid.sync();
	} while(!fp_stack[0][FP_READ(iter_idx[0])].load(cuda::memory_order_relaxed));


	TOGGLE_STEP_PARITY(Node);
	nrof_instances = node->nrof_instances2(STEP_PARITY(Node));
	executeStep<print_Node>(nrof_instances, grid, block, &stable);
	node->update_counters(!STEP_PARITY(Node));
}


int main(int argc, char **argv) {
	if (argc != 2) {
		printf("Supply a .init file.\n");
		exit(1);
	}

	std::vector<InitFile::StructInfo> structs = InitFile::parse(argv[1]);
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 4194304);
	CHECK(hipHostRegister(&host_Edge, sizeof(Edge), hipHostRegisterDefault));
	CHECK(hipHostRegister(&host_Node, sizeof(Node), hipHostRegisterDefault));
	CHECK(hipHostRegister(&host_NodeSet, sizeof(NodeSet), hipHostRegisterDefault));

	host_Edge.initialise(&structs[0], 10000);
	host_Node.initialise(&structs[1], 10000);
	host_NodeSet.initialise(&structs[2], 10000);

	CHECK(hipDeviceSynchronize());

	Edge * const loc_edge = (Edge*)host_Edge.to_device();
	Node * const loc_node = (Node*)host_Node.to_device();
	NodeSet * const loc_nodeset = (NodeSet*)host_NodeSet.to_device();

	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(edge), &loc_edge, sizeof(Edge * const)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(node), &loc_node, sizeof(Node * const)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(nodeset), &loc_nodeset, sizeof(NodeSet * const)));

	cuda::atomic<bool, cuda::thread_scope_device>* fp_stack_address;
	hipGetSymbolAddress((void **)&fp_stack_address, fp_stack);
	CHECK(hipMemset((void*)fp_stack_address, 1, FP_DEPTH * 3 * sizeof(cuda::atomic<bool, cuda::thread_scope_device>)));
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);


	void* schedule_kernel_args[] = {};
	auto dims = ADL::get_launch_dims(1875, (void*)schedule_kernel);

	CHECK(
		hipLaunchCooperativeKernel(
			(void*)schedule_kernel,
			std::get<0>(dims),
			std::get<1>(dims),
			schedule_kernel_args
		)
	);
	CHECK(hipDeviceSynchronize());


	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	printf("Total walltime: %0.2f ms\n");

}
