#include "hip/hip_runtime.h"
#include "ADL.h"
#include "Struct.h"
#include "fp_manager.h"
#include "init_file.h"
#include <stdio.h>
#include <vector>


#define FP_DEPTH 2
#define SET_PARAM(P, V, T, I) ({if (I != 0) { T read_val = P; T write_val = V; if (read_val != write_val) {P = write_val; FP->set();}}})


class NodeSet : public Struct {
public:
	NodeSet (void) : Struct() {}
	
	union {
		void* parameters[5];
		struct {
			ADL::RefType* pivot;
			ADL::BoolType* scc;
			ADL::RefType* f_and_b;
			ADL::RefType* not_f_and_b;
			ADL::RefType* f_and_not_b;
		};
	};

	void assertCorrectInfo(InitFile::StructInfo* info) {
		assert (info->name == "NodeSet");
		assert (info->parameter_types.size() == 5);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Bool);
		assert (info->parameter_types[2] == ADL::Ref);
		assert (info->parameter_types[3] == ADL::Ref);
		assert (info->parameter_types[4] == ADL::Ref);
	};

	void** get_parameters(void) {
		return parameters;
	}

	size_t child_size(void) {
		return sizeof(NodeSet);
	}

	__host__ __device__ RefType create_instance(ADL::RefType _pivot, ADL::BoolType _scc, ADL::RefType _f_and_b, ADL::RefType _not_f_and_b, ADL::RefType _f_and_not_b) {
		RefType slot = claim_instance();
		pivot[slot] = _pivot;
		scc[slot] = _scc;
		f_and_b[slot] = _f_and_b;
		not_f_and_b[slot] = _not_f_and_b;
		f_and_not_b[slot] = _f_and_not_b;
		return slot;
	}
};

class Node : public Struct {
public:
	Node (void) : Struct() {}
	
	union {
		void* parameters[3];
		struct {
			ADL::RefType* set;
			ADL::BoolType* fwd;
			ADL::BoolType* bwd;
		};
	};

	void assertCorrectInfo(InitFile::StructInfo* info) {
		assert (info->name == "Node");
		assert (info->parameter_types.size() == 3);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Bool);
		assert (info->parameter_types[2] == ADL::Bool);
	};

	void** get_parameters(void) {
		return parameters;
	}

	size_t child_size(void) {
		return sizeof(Node);
	}

	__host__ __device__ RefType create_instance(ADL::RefType _set, ADL::BoolType _fwd, ADL::BoolType _bwd) {
		RefType slot = claim_instance();
		set[slot] = _set;
		fwd[slot] = _fwd;
		bwd[slot] = _bwd;
		return slot;
	}
};

class Edge : public Struct {
public:
	Edge (void) : Struct() {}
	
	union {
		void* parameters[2];
		struct {
			ADL::RefType* s;
			ADL::RefType* t;
		};
	};

	void assertCorrectInfo(InitFile::StructInfo* info) {
		assert (info->name == "Edge");
		assert (info->parameter_types.size() == 2);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Ref);
	};

	void** get_parameters(void) {
		return parameters;
	}

	size_t child_size(void) {
		return sizeof(Edge);
	}

	__host__ __device__ RefType create_instance(ADL::RefType _s, ADL::RefType _t) {
		RefType slot = claim_instance();
		s[slot] = _s;
		t[slot] = _t;
		return slot;
	}
};








__global__ void NodeSet_divide_into_sets_reset_pivot(FPManager* FP, NodeSet* const nodeset, Node* const node, Edge* const edge){
	grid_group grid = this_grid();
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!nodeset->is_active(self)) { return; }

	SET_PARAM(node->pivot[self], 0, RefType, self);
}

__global__ void NodeSet_pivot_win_allocate_sets(FPManager* FP, NodeSet* const nodeset, Node* const node, Edge* const edge, NodeSet* const host_nodeset){
	grid_group grid = this_grid();
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!nodeset->is_active(self)) { return; }

	if (node->pivot[self] == 0) {
		SET_PARAM(bool->scc[self], true, BoolType, self);
	}
	if (node->pivot[self] != 0) {
		SET_PARAM(bool->fwd[node->pivot[self]], true, BoolType, node->pivot[self]);
		SET_PARAM(bool->bwd[node->pivot[self]], true, BoolType, node->pivot[self]);
	}
	if (!bool->scc[self]) {
		RefType intermediate = nodeset->create_instance(0, false, 0, 0, 0);
		bool->scc[intermediate] = true;
		SET_PARAM(nodeset->f_and_b[self], intermediate, RefType, self);
		SET_PARAM(nodeset->f_and_not_b[self], nodeset->create_instance(0, false, 0, 0, 0), RefType, self);
		SET_PARAM(nodeset->not_f_and_b[self], nodeset->create_instance(0, false, 0, 0, 0), RefType, self);
	}

	grid.sync();

	if (self == 0) {
		nodeset->sync_nrof_instances(host_nodeset);
	}

}

__global__ void Node_pivot_nominate(FPManager* FP, NodeSet* const nodeset, Node* const node, Edge* const edge){
	grid_group grid = this_grid();
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!node->is_active(self)) { return; }

	if (!bool->scc[nodeset->set[self]]) {
		SET_PARAM(node->pivot[nodeset->set[self]], self, RefType, nodeset->set[self]);
	}
}

__global__ void Node_divide_into_sets_reset_pivot(FPManager* FP, NodeSet* const nodeset, Node* const node, Edge* const edge){
	grid_group grid = this_grid();
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!node->is_active(self)) { return; }

	if (bool->fwd[self] && bool->bwd[self]) {
		SET_PARAM(nodeset->set[self], nodeset->f_and_b[nodeset->set[self]], RefType, self);
	}
	if ((!bool->fwd[self]) && bool->bwd[self]) {
		SET_PARAM(nodeset->set[self], nodeset->not_f_and_b[nodeset->set[self]], RefType, self);
	}
	if (bool->fwd[self] && (!bool->bwd[self])) {
		SET_PARAM(nodeset->set[self], nodeset->f_and_not_b[nodeset->set[self]], RefType, self);
	}
	SET_PARAM(bool->fwd[self], false, BoolType, self);
	SET_PARAM(bool->bwd[self], false, BoolType, self);
}

__global__ void Edge_compute_fwd_bwd(FPManager* FP, NodeSet* const nodeset, Node* const node, Edge* const edge){
	grid_group grid = this_grid();
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!edge->is_active(self)) { return; }

	if (nodeset->set[node->t[self]] == nodeset->set[node->s[self]]) {
		if bool->fwd[node->s[self]] {
			SET_PARAM(bool->fwd[node->t[self]], true, BoolType, node->t[self]);
		}
		if bool->bwd[node->t[self]] {
			SET_PARAM(bool->bwd[node->s[self]], true, BoolType, node->s[self]);
		}
	}
}


int main(int argc, char **argv) {
	if (argc != 2) {
		printf("Supply a .init file.\n");
		exit(1);
	}
	
	std::vector<InitFile::StructInfo> structs = InitFile::parse(argv[1]);
	Edge host_Edge = Edge();
	Node host_Node = Node();
	NodeSet host_NodeSet = NodeSet();

	host_Edge.initialise(&structs[0], 100);
	host_Node.initialise(&structs[1], 100);
	host_NodeSet.initialise(&structs[2], 100);

	CHECK(hipDeviceSynchronize());

	Edge* gm_Edge = (Edge*)host_Edge.to_device();
	Node* gm_Node = (Node*)host_Node.to_device();
	NodeSet* gm_NodeSet = (NodeSet*)host_NodeSet.to_device();



	FPManager host_FP = FPManager(FP_DEPTH); // initially not done
	FPManager* device_FP = host_FP.to_device();

	host_FP.push();
	do{
		host_FP.reset();
		host_FP.copy_to(device_FP);
		void* Node_pivot_nominate_args[] = {
	,
			
};
		CHECK(
			hipLaunchCooperativeKernel(
				(void*)Node_pivot_nominate,
				(host_Node.nrof_instances() + 512 - 1)/512,
				512,
				Node_pivot_nominate_args
			)
		);
		CHECK(hipDeviceSynchronize());

		void* NodeSet_pivot_win_allocate_sets_args[] = {
	,
			
};
		CHECK(
			hipLaunchCooperativeKernel(
				(void*)NodeSet_pivot_win_allocate_sets,
				(host_NodeSet.nrof_instances() + 512 - 1)/512,
				512,
				NodeSet_pivot_win_allocate_sets_args
			)
		);
		CHECK(hipDeviceSynchronize());

		host_FP.push();
		do{
			host_FP.reset();
			host_FP.copy_to(device_FP);
			void* Edge_compute_fwd_bwd_args[] = {
	,
				
};
			CHECK(
				hipLaunchCooperativeKernel(
					(void*)Edge_compute_fwd_bwd,
					(host_Edge.nrof_instances() + 512 - 1)/512,
					512,
					Edge_compute_fwd_bwd_args
				)
			);
			CHECK(hipDeviceSynchronize());

			host_FP.copy_from(device_FP);
			if(!host_FP.done()) host_FP.clear();
		}
		while(!host_FP.done());
		host_FP.pop();

		void* NodeSet_divide_into_sets_reset_pivot_args[] = {
	,
			
};
		CHECK(
			hipLaunchCooperativeKernel(
				(void*)NodeSet_divide_into_sets_reset_pivot,
				(host_NodeSet.nrof_instances() + 512 - 1)/512,
				512,
				NodeSet_divide_into_sets_reset_pivot_args
			)
		);
		CHECK(hipDeviceSynchronize());
		void* Node_divide_into_sets_reset_pivot_args[] = {
	,
			
};
		CHECK(
			hipLaunchCooperativeKernel(
				(void*)Node_divide_into_sets_reset_pivot,
				(host_Node.nrof_instances() + 512 - 1)/512,
				512,
				Node_divide_into_sets_reset_pivot_args
			)
		);
		CHECK(hipDeviceSynchronize());

		host_FP.copy_from(device_FP);
		if(!host_FP.done()) host_FP.clear();
	}
	while(!host_FP.done());
	host_FP.pop();



	
}
