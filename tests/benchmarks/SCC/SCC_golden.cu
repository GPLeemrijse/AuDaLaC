#include "hip/hip_runtime.h"
#include "ADL.h"
#include "Struct.h"
#include "fp_manager.h"
#include "init_file.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;


#define FP_DEPTH 2
#define SET_PARAM(P, V, T, I) ({if (I != 0) { T read_val = P; T write_val = V; if (read_val != write_val) {P = write_val; FP->clear();}}})


class NodeSet : public Struct {
public:
	NodeSet (void) : Struct() {}
	
	union {
		void* parameters[5];
		struct {
			ADL::RefType* pivot;
			ADL::BoolType* scc;
			ADL::RefType* f_and_b;
			ADL::RefType* not_f_and_b;
			ADL::RefType* f_and_not_b;
		};
	};

	void assertCorrectInfo(InitFile::StructInfo* info) {
		assert (info->name == "NodeSet");
		assert (info->parameter_types.size() == 5);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Bool);
		assert (info->parameter_types[2] == ADL::Ref);
		assert (info->parameter_types[3] == ADL::Ref);
		assert (info->parameter_types[4] == ADL::Ref);
	};

	void** get_parameters(void) {
		return parameters;
	}

	size_t child_size(void) {
		return sizeof(NodeSet);
	}

	__host__ __device__ RefType create_instance(ADL::RefType _pivot, ADL::BoolType _scc, ADL::RefType _f_and_b, ADL::RefType _not_f_and_b, ADL::RefType _f_and_not_b) {
		RefType slot = claim_instance();
		pivot[slot] = _pivot;
		scc[slot] = _scc;
		f_and_b[slot] = _f_and_b;
		not_f_and_b[slot] = _not_f_and_b;
		f_and_not_b[slot] = _f_and_not_b;
		return slot;
	}
};

class Node : public Struct {
public:
	Node (void) : Struct() {}
	
	union {
		void* parameters[3];
		struct {
			ADL::RefType* set;
			ADL::BoolType* fwd;
			ADL::BoolType* bwd;
		};
	};

	void assertCorrectInfo(InitFile::StructInfo* info) {
		assert (info->name == "Node");
		assert (info->parameter_types.size() == 3);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Bool);
		assert (info->parameter_types[2] == ADL::Bool);
	};

	void** get_parameters(void) {
		return parameters;
	}

	size_t child_size(void) {
		return sizeof(Node);
	}

	__host__ __device__ RefType create_instance(ADL::RefType _set, ADL::BoolType _fwd, ADL::BoolType _bwd) {
		RefType slot = claim_instance();
		set[slot] = _set;
		fwd[slot] = _fwd;
		bwd[slot] = _bwd;
		return slot;
	}
};

class Edge : public Struct {
public:
	Edge (void) : Struct() {}
	
	union {
		void* parameters[2];
		struct {
			ADL::RefType* s;
			ADL::RefType* t;
		};
	};

	void assertCorrectInfo(InitFile::StructInfo* info) {
		assert (info->name == "Edge");
		assert (info->parameter_types.size() == 2);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Ref);
	};

	void** get_parameters(void) {
		return parameters;
	}

	size_t child_size(void) {
		return sizeof(Edge);
	}

	__host__ __device__ RefType create_instance(ADL::RefType _s, ADL::RefType _t) {
		RefType slot = claim_instance();
		s[slot] = _s;
		t[slot] = _t;
		return slot;
	}
};







__global__ void NodeSet_divide_into_sets_reset_pivot(FPManager* FP, NodeSet * const nodeset, Node * const node, Edge * const edge){
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!nodeset->is_active(self)) { return; }

	SET_PARAM((nodeset->pivot[self]), 0, RefType, self);
}

__global__ void NodeSet_pivot_win_allocate_sets(FPManager* FP,
												NodeSet * const nodeset,
												NodeSet * const host_nodeset,
												Node * const node,
												Edge * const edge){
	grid_group grid = this_grid();

	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!nodeset->is_active(self)) { grid.sync(); return; }

	if ((nodeset->pivot[self]) == 0) {
		SET_PARAM((nodeset->scc[self]), true, BoolType, self);
	}
	if ((nodeset->pivot[self]) != 0) {
		SET_PARAM((node->fwd[nodeset->pivot[self]]), true, BoolType, (nodeset->pivot[self]));
		SET_PARAM((node->bwd[nodeset->pivot[self]]), true, BoolType, (nodeset->pivot[self]));
	}
	if (!(nodeset->scc[self])) {
		RefType intermediate = nodeset->create_instance(0, false, 0, 0, 0);
		(nodeset->scc[intermediate]) = true;
		SET_PARAM((nodeset->f_and_b[self]), (intermediate), RefType, self);
		SET_PARAM((nodeset->f_and_not_b[self]), nodeset->create_instance(0, false, 0, 0, 0), RefType, self);
		SET_PARAM((nodeset->not_f_and_b[self]), nodeset->create_instance(0, false, 0, 0, 0), RefType, self);
	}

	grid.sync();

	if (self == 0){
		nodeset->sync_nrof_instances(host_nodeset);
	}
}

__global__ void Node_pivot_nominate(FPManager* FP, NodeSet * const nodeset, Node * const node, Edge * const edge){
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!node->is_active(self)) { return; }

	if (!(nodeset->scc[node->set[self]])) {
		SET_PARAM((nodeset->pivot[node->set[self]]), self, RefType, (node->set[self]));
	}
}

__global__ void Node_divide_into_sets_reset_pivot(FPManager* FP, NodeSet * const nodeset, Node * const node, Edge * const edge){
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!node->is_active(self)) { return; }

	if ((node->fwd[self]) && (node->bwd[self])) {
		SET_PARAM((node->set[self]), (nodeset->f_and_b[node->set[self]]), RefType, self);
	}
	if ((!(node->fwd[self])) && (node->bwd[self])) {
		SET_PARAM((node->set[self]), (nodeset->not_f_and_b[node->set[self]]), RefType, self);
	}
	if ((node->fwd[self]) && (!(node->bwd[self]))) {
		SET_PARAM((node->set[self]), (nodeset->f_and_not_b[node->set[self]]), RefType, self);
	}
	SET_PARAM((node->fwd[self]), false, BoolType, self);
	SET_PARAM((node->bwd[self]), false, BoolType, self);
}

__global__ void Edge_compute_fwd_bwd(FPManager* FP, NodeSet * const nodeset, Node * const node, Edge * const edge){
	RefType self = blockDim.x * blockIdx.x + threadIdx.x;
	if(!edge->is_active(self)) { return; }

	if ((node->set[edge->t[self]]) == (node->set[edge->s[self]])) {
		if (node->fwd[edge->s[self]]) {
			SET_PARAM((node->fwd[edge->t[self]]), true, BoolType, (edge->t[self]));
		}
		if (node->bwd[edge->t[self]]) {
			SET_PARAM((node->bwd[edge->s[self]]), true, BoolType, (edge->s[self]));
		}
	}
}


int main(int argc, char **argv) {
	if (argc != 2) {
		printf("Supply a .init file.\n");
		exit(1);
	}
	
	std::vector<InitFile::StructInfo> structs = InitFile::parse(argv[1]);
	Edge host_Edge = Edge();
	Node host_Node = Node();
	NodeSet host_NodeSet = NodeSet();
	NodeSet* host_NodeSet_ptr = &host_NodeSet;
	CHECK(hipHostRegister(&host_NodeSet, sizeof(NodeSet), hipHostRegisterDefault));
	CHECK(hipHostRegister(&host_Node, sizeof(Node), hipHostRegisterDefault));
	CHECK(hipHostRegister(&host_Edge, sizeof(Edge), hipHostRegisterDefault));

	host_Edge.initialise(&structs[0], 100);
	host_Node.initialise(&structs[1], 100);
	host_NodeSet.initialise(&structs[2], 100);

	CHECK(hipDeviceSynchronize());

	Edge* gm_Edge = (Edge*)host_Edge.to_device();
	Node* gm_Node = (Node*)host_Node.to_device();
	NodeSet* gm_NodeSet = (NodeSet*)host_NodeSet.to_device();



	FPManager host_FP = FPManager(FP_DEPTH); // initially not done
	FPManager* device_FP = host_FP.to_device();

	host_FP.push();
	do{
		host_FP.reset();
		host_FP.copy_to(device_FP);
		Node_pivot_nominate<<<(host_Node.nrof_instances() + 512 - 1)/512, 512>>>(device_FP, gm_NodeSet, gm_Node, gm_Edge);
		CHECK(hipDeviceSynchronize());

		void* args[] = {
			(void*)&device_FP,
			(void*)&gm_NodeSet,
			(void*)&host_NodeSet_ptr,
			(void*)&gm_Node,
			(void*)&gm_Edge
		};

		CHECK(
			hipLaunchCooperativeKernel(
				(void*)NodeSet_pivot_win_allocate_sets,
				(host_NodeSet.nrof_instances() + 512 - 1)/512,
				512,
				args
			)
		);
		CHECK(hipDeviceSynchronize());

		host_FP.push();
		do{
			host_FP.reset();
			host_FP.copy_to(device_FP);
			Edge_compute_fwd_bwd<<<(host_Edge.nrof_instances() + 512 - 1)/512, 512>>>(device_FP, gm_NodeSet, gm_Node, gm_Edge);
			CHECK(hipDeviceSynchronize());


			host_FP.copy_from(device_FP);
		}
		while(!host_FP.done());
		host_FP.pop();

		NodeSet_divide_into_sets_reset_pivot<<<(host_NodeSet.nrof_instances() + 512 - 1)/512, 512>>>(device_FP, gm_NodeSet, gm_Node, gm_Edge);
		CHECK(hipDeviceSynchronize());
		Node_divide_into_sets_reset_pivot<<<(host_Node.nrof_instances() + 512 - 1)/512, 512>>>(device_FP, gm_NodeSet, gm_Node, gm_Edge);
		CHECK(hipDeviceSynchronize());


		host_FP.copy_from(device_FP);
	}
	while(!host_FP.done());
	host_FP.pop();


	
}
