#include "hip/hip_runtime.h"
#define I_PER_THREAD 140
#define THREADS_PER_BLOCK 256
#define ATOMIC(T) cuda::atomic<T, cuda::thread_scope_device>
#define STORE(A, B) A.store(B, cuda::memory_order_relaxed)
#define LOAD(A) A.load(cuda::memory_order_relaxed)
#define Node_MASK (1ULL << 0)
#define Edge_MASK (1ULL << 1)
#define Measure_MASK (1ULL << 2)
#define STEP_PARITY(STRUCT) ((bool)(struct_step_parity & STRUCT ## _MASK))
#define TOGGLE_STEP_PARITY(STRUCT) {struct_step_parity ^= STRUCT ## _MASK;}


#include "ADL.h"
#include "Struct.h"
#include "init_file.h"
#include <hip/hip_cooperative_groups.h>
#include <cuda/atomic>
#include <stdio.h>
#include <vector>


class Node : public Struct {
public:
	Node (void) : Struct() {}
	
	ATOMIC(NatType)* p;
	ATOMIC(BoolType)* owner;
	ATOMIC(RefType)* rho;
	ATOMIC(RefType)* candidate;
	ATOMIC(RefType)* max;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "Node");
		assert (info->parameter_types.size() == 5);
		assert (info->parameter_types[0] == ADL::Nat);
		assert (info->parameter_types[1] == ADL::Bool);
		assert (info->parameter_types[2] == ADL::Ref);
		assert (info->parameter_types[3] == ADL::Ref);
		assert (info->parameter_types[4] == ADL::Ref);
	};

	void** get_parameters(void) {
		return (void**)&p;
	}

	size_t child_size(void) {
		return sizeof(Node);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[5] = {
			sizeof(ATOMIC(NatType)),
			sizeof(ATOMIC(BoolType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(NatType _p,
									   BoolType _owner,
									   RefType _rho,
									   RefType _candidate,
									   RefType _max,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(p[slot], _p);
		STORE(owner[slot], _owner);
		STORE(rho[slot], _rho);
		STORE(candidate[slot], _candidate);
		STORE(max[slot], _max);
		*stable = false;
		return slot;
	}
};

class Edge : public Struct {
public:
	Edge (void) : Struct() {}
	
	ATOMIC(RefType)* v;
	ATOMIC(RefType)* w;
	ATOMIC(RefType)* m;
	ATOMIC(RefType)* max;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "Edge");
		assert (info->parameter_types.size() == 4);
		assert (info->parameter_types[0] == ADL::Ref);
		assert (info->parameter_types[1] == ADL::Ref);
		assert (info->parameter_types[2] == ADL::Ref);
		assert (info->parameter_types[3] == ADL::Ref);
	};

	void** get_parameters(void) {
		return (void**)&v;
	}

	size_t child_size(void) {
		return sizeof(Edge);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[4] = {
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType)),
			sizeof(ATOMIC(RefType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(RefType _v,
									   RefType _w,
									   RefType _m,
									   RefType _max,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(v[slot], _v);
		STORE(w[slot], _w);
		STORE(m[slot], _m);
		STORE(max[slot], _max);
		*stable = false;
		return slot;
	}
};

class Measure : public Struct {
public:
	Measure (void) : Struct() {}
	
	ATOMIC(BoolType)* top;
	ATOMIC(NatType)* p1;
	ATOMIC(NatType)* p3;

	void assert_correct_info(InitFile::StructInfo* info) {
		assert (info->name == "Measure");
		assert (info->parameter_types.size() == 3);
		assert (info->parameter_types[0] == ADL::Bool);
		assert (info->parameter_types[1] == ADL::Nat);
		assert (info->parameter_types[2] == ADL::Nat);
	};

	void** get_parameters(void) {
		return (void**)&top;
	}

	size_t child_size(void) {
		return sizeof(Measure);
	}

	size_t param_size(uint idx) {
		static const size_t sizes[3] = {
			sizeof(ATOMIC(BoolType)),
			sizeof(ATOMIC(NatType)),
			sizeof(ATOMIC(NatType))
		};
		return sizes[idx];
	}

	__device__ RefType create_instance(BoolType _top,
									   NatType _p1,
									   NatType _p3,
									   bool* stable){
		RefType slot = claim_instance2();
		STORE(top[slot], _top);
		STORE(p1[slot], _p1);
		STORE(p3[slot], _p3);
		*stable = false;
		return slot;
	}
};

using namespace cooperative_groups;

Edge host_Edge = Edge();
Measure host_Measure = Measure();
Node host_Node = Node();

Edge* host_Edge_ptr = &host_Edge;
Measure* host_Measure_ptr = &host_Measure;
Node* host_Node_ptr = &host_Node;

__device__ Edge* __restrict__ edge;
__device__ Measure* __restrict__ measure;
__device__ Node* __restrict__ node;

__device__ uint nrof_odd_wins = 0;
#define FP_DEPTH 2
/* Transform an iter_idx into the fp_stack index
   associated with that operation.
*/
#define FP_SET(X) (X)
#define FP_RESET(X) ((X) + 1 >= 3 ? (X) + 1 - 3 : (X) + 1)
#define FP_READ(X) ((X) + 2 >= 3 ? (X) + 2 - 3 : (X) + 2)

__device__ cuda::atomic<bool, cuda::thread_scope_device> fp_stack[FP_DEPTH][3];

__device__ void clear_stack(int lvl, uint8_t* iter_idx) {
	/*	Clears the stack on the FP_SET side.
		The FP_RESET and FP_READ sides should remain the same.
	*/
	while(lvl >= 0){
		fp_stack[lvl][FP_SET(iter_idx[lvl])].store(false, cuda::memory_order_relaxed);
		lvl--;
	}
}

typedef void(*step_func)(RefType, bool*);
template <step_func Step>
__device__ void executeStep(inst_size nrof_instances, grid_group grid, thread_block block, bool* stable){
	for(int i = 0; i < I_PER_THREAD; i++){
		const RefType self = block.size() * (i + grid.block_rank() * I_PER_THREAD) + block.thread_rank();
		if (self >= nrof_instances) break;

		Step(self, stable);
	}
}
template<typename T>
__device__ void SetParam(const RefType owner, ATOMIC(T) * const params, const T new_val, bool* stable) {
	if (owner != 0){
		T old_val = LOAD(params[owner]);
		if (old_val != new_val){
			STORE(params[owner], new_val);
			*stable = false;
		}
	}
}

__device__ void Node_print(const RefType self,
						   bool* stable){
	if (self != 0) {
		printf("Node(%u): p=%u, owner=%u, rho=%u, candidate=%u, max=%u\n", self, LOAD(node->p[self]), LOAD(node->owner[self]), LOAD(node->rho[self]), LOAD(node->candidate[self]), LOAD(node->max[self]));
	}
}

__device__ void Node_max_candidate(const RefType self,
								   bool* stable){
	
	if (((LOAD(node->candidate[self]) != 0) && (!LOAD(measure->top[LOAD(node->rho[self])])))) {
		BoolType copy = ((LOAD(measure->top[LOAD(node->candidate[self])]) || (LOAD(measure->p1[LOAD(node->candidate[self])]) > LOAD(measure->p1[LOAD(node->rho[self])]))) || ((LOAD(measure->p1[LOAD(node->candidate[self])]) <= LOAD(measure->p1[LOAD(node->rho[self])])) && (LOAD(measure->p3[LOAD(node->candidate[self])]) > LOAD(measure->p3[LOAD(node->rho[self])]))));
		if (copy) {
			// rho.top := candidate.top;
			SetParam<BoolType>(LOAD(node->rho[self]), measure->top, LOAD(measure->top[LOAD(node->candidate[self])]), stable);
			// rho.p1 := candidate.p1;
			SetParam<NatType>(LOAD(node->rho[self]), measure->p1, LOAD(measure->p1[LOAD(node->candidate[self])]), stable);
			// rho.p3 := candidate.p3;
			SetParam<NatType>(LOAD(node->rho[self]), measure->p3, LOAD(measure->p3[LOAD(node->candidate[self])]), stable);
		}
	}
}

__device__ void Node_count_odd(const RefType self,
							   bool* stable){
	
	if (((self != 0) && LOAD(measure->top[LOAD(node->rho[self])]))) {
		atomicInc(&nrof_odd_wins, 0xffffffff);
	}
}

__device__ void Node_print_odd(const RefType self,
							   bool* stable){
	
	if ((self == 0)) {
		printf("Number of odd won vertices = %u\n", nrof_odd_wins);
	}
}

__device__ void Edge_print(const RefType self,
						   bool* stable){
	if (self != 0) {
		printf("Edge(%u): v=%u, w=%u, m=%u, max=%u\n", self, LOAD(edge->v[self]), LOAD(edge->w[self]), LOAD(edge->m[self]), LOAD(edge->max[self]));
	}
}

__device__ void Edge_prog(const RefType self,
						  bool* stable){
	
	BoolType top_old = LOAD(measure->top[LOAD(edge->m[self])]);
	NatType p1_old = LOAD(measure->p1[LOAD(edge->m[self])]);
	NatType p3_old = LOAD(measure->p3[LOAD(edge->m[self])]);
	if (((LOAD(node->p[LOAD(edge->v[self])]) % 2) == 0)) {
		// m.top := w.rho.top;
		SetParam<BoolType>(LOAD(edge->m[self]), measure->top, LOAD(measure->top[LOAD(node->rho[LOAD(edge->w[self])])]), stable);
		if ((!LOAD(measure->top[LOAD(edge->m[self])]))) {
			if ((LOAD(node->p[LOAD(edge->v[self])]) >= 1)) {
				// m.p1 := w.rho.p1;
				SetParam<NatType>(LOAD(edge->m[self]), measure->p1, LOAD(measure->p1[LOAD(node->rho[LOAD(edge->w[self])])]), stable);
			}
			if ((LOAD(node->p[LOAD(edge->v[self])]) < 1)) {
				// m.p1 := 0;
				SetParam<NatType>(LOAD(edge->m[self]), measure->p1, 0, stable);
			}
			if ((LOAD(node->p[LOAD(edge->v[self])]) >= 3)) {
				// m.p3 := w.rho.p3;
				SetParam<NatType>(LOAD(edge->m[self]), measure->p3, LOAD(measure->p3[LOAD(node->rho[LOAD(edge->w[self])])]), stable);
			}
			if ((LOAD(node->p[LOAD(edge->v[self])]) < 3)) {
				// m.p3 := 0;
				SetParam<NatType>(LOAD(edge->m[self]), measure->p3, 0, stable);
			}
		}
	}
	if (((LOAD(node->p[LOAD(edge->v[self])]) % 2) == 1)) {
		if (LOAD(measure->top[LOAD(node->rho[LOAD(edge->w[self])])])) {
			// m.top := true;
			SetParam<BoolType>(LOAD(edge->m[self]), measure->top, true, stable);
		}
		if ((!LOAD(measure->top[LOAD(edge->m[self])]))) {
			BoolType incr = false;
			if (((LOAD(node->p[LOAD(edge->v[self])]) >= 3) && (LOAD(measure->p3[LOAD(node->rho[LOAD(edge->w[self])])]) < LOAD(measure->p3[LOAD(edge->max[self])])))) {
				// m.p3 := w.rho.p3 + 1;
				SetParam<NatType>(LOAD(edge->m[self]), measure->p3, (LOAD(measure->p3[LOAD(node->rho[LOAD(edge->w[self])])]) + 1), stable);
				incr = true;
			}
			if ((!incr)) {
				// m.p3 := 0;
				SetParam<NatType>(LOAD(edge->m[self]), measure->p3, 0, stable);
				if (((LOAD(node->p[LOAD(edge->v[self])]) >= 1) && (LOAD(measure->p1[LOAD(node->rho[LOAD(edge->w[self])])]) < LOAD(measure->p1[LOAD(edge->max[self])])))) {
					// m.p1 := w.rho.p1 + 1;
					SetParam<NatType>(LOAD(edge->m[self]), measure->p1, (LOAD(measure->p1[LOAD(node->rho[LOAD(edge->w[self])])]) + 1), stable);
					incr = true;
				}
				if ((!incr)) {
					// m.top := true;
					SetParam<BoolType>(LOAD(edge->m[self]), measure->top, true, stable);
				}
			}
		}
	}
	BoolType is_stable = (((top_old == LOAD(measure->top[LOAD(edge->m[self])])) && (p1_old == LOAD(measure->p1[LOAD(edge->m[self])]))) && (p3_old == LOAD(measure->p3[LOAD(edge->m[self])])));
	if (((!is_stable) && (LOAD(node->candidate[LOAD(edge->v[self])]) == LOAD(edge->m[self])))) {
		// v.candidate := null;
		SetParam<RefType>(LOAD(edge->v[self]), node->candidate, 0, stable);
	}
}

__device__ void Edge_top(const RefType self,
						 bool* stable){
	
	if (((!LOAD(node->owner[LOAD(edge->v[self])])) && ((LOAD(node->candidate[LOAD(edge->v[self])]) == 0) || ((!LOAD(measure->top[LOAD(edge->m[self])])) && LOAD(measure->top[LOAD(node->candidate[LOAD(edge->v[self])])]))))) {
		// v.candidate := m;
		SetParam<RefType>(LOAD(edge->v[self]), node->candidate, LOAD(edge->m[self]), stable);
	}
	if ((LOAD(node->owner[LOAD(edge->v[self])]) && ((LOAD(node->candidate[LOAD(edge->v[self])]) == 0) || (LOAD(measure->top[LOAD(edge->m[self])]) && (!LOAD(measure->top[LOAD(node->candidate[LOAD(edge->v[self])])])))))) {
		// v.candidate := m;
		SetParam<RefType>(LOAD(edge->v[self]), node->candidate, LOAD(edge->m[self]), stable);
	}
}

__device__ void Edge_priority_1(const RefType self,
								bool* stable){
	
	if ((LOAD(measure->top[LOAD(node->candidate[LOAD(edge->v[self])])]) == LOAD(measure->top[LOAD(edge->m[self])]))) {
		if ((((!LOAD(node->owner[LOAD(edge->v[self])])) && (LOAD(measure->p1[LOAD(edge->m[self])]) < LOAD(measure->p1[LOAD(node->candidate[LOAD(edge->v[self])])]))) || (LOAD(node->owner[LOAD(edge->v[self])]) && (LOAD(measure->p1[LOAD(edge->m[self])]) > LOAD(measure->p1[LOAD(node->candidate[LOAD(edge->v[self])])]))))) {
			// v.candidate := m;
			SetParam<RefType>(LOAD(edge->v[self]), node->candidate, LOAD(edge->m[self]), stable);
		}
	}
}

__device__ void Edge_priority_3(const RefType self,
								bool* stable){
	
	if (((LOAD(measure->top[LOAD(node->candidate[LOAD(edge->v[self])])]) == LOAD(measure->top[LOAD(edge->m[self])])) && (LOAD(measure->p1[LOAD(node->candidate[LOAD(edge->v[self])])]) == LOAD(measure->p1[LOAD(edge->m[self])])))) {
		if ((((!LOAD(node->owner[LOAD(edge->v[self])])) && (LOAD(measure->p3[LOAD(edge->m[self])]) < LOAD(measure->p3[LOAD(node->candidate[LOAD(edge->v[self])])]))) || (LOAD(node->owner[LOAD(edge->v[self])]) && (LOAD(measure->p3[LOAD(edge->m[self])]) > LOAD(measure->p3[LOAD(node->candidate[LOAD(edge->v[self])])]))))) {
			// v.candidate := m;
			SetParam<RefType>(LOAD(edge->v[self]), node->candidate, LOAD(edge->m[self]), stable);
		}
	}
}

__device__ void Edge_self_loops_to_top(const RefType self,
									   bool* stable){
	
	if ((((LOAD(edge->v[self]) == LOAD(edge->w[self])) && LOAD(node->owner[LOAD(edge->v[self])])) && ((LOAD(node->p[LOAD(edge->v[self])]) % 2) == 1))) {
		// v.rho.top := true;
		SetParam<BoolType>(LOAD(node->rho[LOAD(edge->v[self])]), measure->top, true, stable);
	}
}

__device__ void Measure_print(const RefType self,
							  bool* stable){
	if (self != 0) {
		printf("Measure(%u): top=%u, p1=%u, p3=%u\n", self, LOAD(measure->top[self]), LOAD(measure->p1[self]), LOAD(measure->p3[self]));
	}
}


__global__ void schedule_kernel(){
	const grid_group grid = this_grid();
	const thread_block block = this_thread_block();
	const bool is_thread0 = grid.thread_rank() == 0;
	inst_size nrof_instances;
	uint64_t struct_step_parity = 0; // bitmask
	bool stable = true; // Only used to compile steps outside fixpoints
	uint8_t iter_idx[FP_DEPTH] = {0}; // Denotes which fp_stack index ([0, 2]) is currently being set.

	TOGGLE_STEP_PARITY(Edge);
	nrof_instances = edge->nrof_instances2(STEP_PARITY(Edge));
	executeStep<Edge_self_loops_to_top>(nrof_instances, grid, block, &stable);
	edge->update_counters(!STEP_PARITY(Edge));

	grid.sync();

	do{
		bool stable = true;
		if (is_thread0){
			/* Resets the next fp_stack index in advance. */
			fp_stack[0][FP_RESET(iter_idx[0])].store(true, cuda::memory_order_relaxed);
		}


		TOGGLE_STEP_PARITY(Edge);
		nrof_instances = edge->nrof_instances2(STEP_PARITY(Edge));
		executeStep<Edge_prog>(nrof_instances, grid, block, &stable);
		edge->update_counters(!STEP_PARITY(Edge));

		grid.sync();

		do{
			bool stable = true;
			if (is_thread0){
				/* Resets the next fp_stack index in advance. */
				fp_stack[1][FP_RESET(iter_idx[1])].store(true, cuda::memory_order_relaxed);
			}


			TOGGLE_STEP_PARITY(Edge);
			nrof_instances = edge->nrof_instances2(STEP_PARITY(Edge));
			executeStep<Edge_top>(nrof_instances, grid, block, &stable);
			edge->update_counters(!STEP_PARITY(Edge));
			if(!stable){
				clear_stack(1, iter_idx);
			}
			/* The next index to set is the one that has been reset. */
			iter_idx[1] = FP_RESET(iter_idx[1]);
			grid.sync();
		} while(!fp_stack[1][FP_READ(iter_idx[1])].load(cuda::memory_order_relaxed));


		do{
			bool stable = true;
			if (is_thread0){
				/* Resets the next fp_stack index in advance. */
				fp_stack[1][FP_RESET(iter_idx[1])].store(true, cuda::memory_order_relaxed);
			}


			TOGGLE_STEP_PARITY(Edge);
			nrof_instances = edge->nrof_instances2(STEP_PARITY(Edge));
			executeStep<Edge_priority_1>(nrof_instances, grid, block, &stable);
			edge->update_counters(!STEP_PARITY(Edge));
			if(!stable){
				clear_stack(1, iter_idx);
			}
			/* The next index to set is the one that has been reset. */
			iter_idx[1] = FP_RESET(iter_idx[1]);
			grid.sync();
		} while(!fp_stack[1][FP_READ(iter_idx[1])].load(cuda::memory_order_relaxed));


		do{
			bool stable = true;
			if (is_thread0){
				/* Resets the next fp_stack index in advance. */
				fp_stack[1][FP_RESET(iter_idx[1])].store(true, cuda::memory_order_relaxed);
			}


			TOGGLE_STEP_PARITY(Edge);
			nrof_instances = edge->nrof_instances2(STEP_PARITY(Edge));
			executeStep<Edge_priority_3>(nrof_instances, grid, block, &stable);
			edge->update_counters(!STEP_PARITY(Edge));
			if(!stable){
				clear_stack(1, iter_idx);
			}
			/* The next index to set is the one that has been reset. */
			iter_idx[1] = FP_RESET(iter_idx[1]);
			grid.sync();
		} while(!fp_stack[1][FP_READ(iter_idx[1])].load(cuda::memory_order_relaxed));


		TOGGLE_STEP_PARITY(Node);
		nrof_instances = node->nrof_instances2(STEP_PARITY(Node));
		executeStep<Node_max_candidate>(nrof_instances, grid, block, &stable);
		node->update_counters(!STEP_PARITY(Node));
		if(!stable){
			clear_stack(0, iter_idx);
		}
		/* The next index to set is the one that has been reset. */
		iter_idx[0] = FP_RESET(iter_idx[0]);
		grid.sync();
	} while(!fp_stack[0][FP_READ(iter_idx[0])].load(cuda::memory_order_relaxed));


	TOGGLE_STEP_PARITY(Node);
	nrof_instances = node->nrof_instances2(STEP_PARITY(Node));
	executeStep<Node_count_odd>(nrof_instances, grid, block, &stable);
	node->update_counters(!STEP_PARITY(Node));

	grid.sync();

	TOGGLE_STEP_PARITY(Node);
	nrof_instances = node->nrof_instances2(STEP_PARITY(Node));
	executeStep<Node_print_odd>(nrof_instances, grid, block, &stable);
	node->update_counters(!STEP_PARITY(Node));
}


int main(int argc, char **argv) {
	if (argc != 2) {
		printf("Supply a .init file.\n");
		exit(1);
	}

	std::vector<InitFile::StructInfo> structs = InitFile::parse(argv[1]);
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 4194304);
	CHECK(hipHostRegister(&host_Edge, sizeof(Edge), hipHostRegisterDefault));
	CHECK(hipHostRegister(&host_Measure, sizeof(Measure), hipHostRegisterDefault));
	CHECK(hipHostRegister(&host_Node, sizeof(Node), hipHostRegisterDefault));

	host_Edge.initialise(&structs[0], 7236208);
	host_Measure.initialise(&structs[1], 7236208);
	host_Node.initialise(&structs[2], 7236208);

	CHECK(hipDeviceSynchronize());

	Edge * const loc_edge = (Edge*)host_Edge.to_device();
	Measure * const loc_measure = (Measure*)host_Measure.to_device();
	Node * const loc_node = (Node*)host_Node.to_device();

	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(edge), &loc_edge, sizeof(Edge * const)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(measure), &loc_measure, sizeof(Measure * const)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(node), &loc_node, sizeof(Node * const)));

	cuda::atomic<bool, cuda::thread_scope_device>* fp_stack_address;
	CHECK(hipGetSymbolAddress((void **)&fp_stack_address, fp_stack));
	CHECK(hipMemset((void*)fp_stack_address, 1, FP_DEPTH * 3 * sizeof(cuda::atomic<bool, cuda::thread_scope_device>)));
	void* schedule_kernel_args[] = {};
	auto dims = ADL::get_launch_dims(51688, (void*)schedule_kernel);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);


	CHECK(
		hipLaunchCooperativeKernel(
			(void*)schedule_kernel,
			std::get<0>(dims),
			std::get<1>(dims),
			schedule_kernel_args
		)
	);


	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	fprintf(stderr, "Total walltime GPU: %0.2f ms\n", ms);

}
