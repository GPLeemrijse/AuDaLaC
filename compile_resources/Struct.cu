#include "hip/hip_runtime.h"
#include "Struct.h"
#include "ADL.h"
#include <assert.h>

__host__ Struct::Struct(void) : is_initialised(false) {}

__host__ void Struct::free(void) {
	if (is_initialised) {
		void** params = get_parameters();

		for (int p = 0; p < nrof_parameters; p++){ 
			CHECK(
				hipFree(params[p])
			);
		}
	}
}

__host__ __device__ inline ADL::RefType Struct::claim_instance(void) {
	#ifdef __CUDA_ARCH__
	    ADL::RefType slot = atomicInc(&instantiated_instances, capacity);
	#else
	    ADL::RefType slot = instantiated_instances++;
	#endif
	return slot;
}

__host__ __device__ bool Struct::is_active(RefType instance){
	return instance < active_instances;
}

__host__ void* Struct::to_device(void) {
	void* device_ptr;
	size_t s = child_size();

	CHECK(
		hipMalloc(&device_ptr, s)
	);

	CHECK(
		hipMemcpy(device_ptr, this, s, hipMemcpyHostToDevice)
	);

	return device_ptr;
}

__host__ void Struct::initialise(InitFile::StructInfo* info, inst_size capacity){
	assertCorrectInfo(info);
	assert (info->nrof_instances < capacity);

	void** params = get_parameters();
	nrof_parameters = info->parameter_data.size();

	for (int p = 0; p < nrof_parameters; p++){
		size_t param_size = size_of_type(info->parameter_types[p]);

		CHECK(
			hipMalloc(&params[p], param_size * capacity)
		);
		
		// Copy initial instances
		CHECK(
			hipMemcpyAsync(
				&((uint8_t*)params[p])[param_size], // free first slot for null instance
				info->parameter_data[p],
				param_size * info->nrof_instances,
				hipMemcpyHostToDevice
			)
		);

		// Copy null-instance
		CHECK(
			hipMemcpyAsync(
				params[p],
				ADL::default_value(info->parameter_types[p]),
				param_size,
				hipMemcpyHostToDevice
			)
		);
	}

	instantiated_instances = info->nrof_instances + 1; // null-instance
	active_instances = info->nrof_instances + 1;
	capacity = capacity;
	is_initialised = true;
}