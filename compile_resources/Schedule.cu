#include "Schedule.h"
#include "ADL.h"

void Schedule::add_step(void* kernel, inst_size capacity) {
	int min_grid_size;
	int dyn_block_size;
	hipOccupancyMaxPotentialBlockSize(&min_grid_size, &dyn_block_size, kernel, 0, 0);

	if (capacity == 1) {
		dyn_block_size = 1;
	}
	dim3 blockDim(dyn_block_size);
	dim3 gridDim((capacity + dyn_block_size - 1) / dyn_block_size);

	k_params.func = kernel;
	k_params.gridDim = gridDim;
	k_params.blockDim = blockDim;
	k_params.sharedMemBytes = 0;
	void* args[1] = {
		(void*)&current->lvl
	};
	k_params.kernelParams = args;
	k_params.extra = NULL;

	hipGraphNode_t new_node;
	if (current->last_node == NULL){
		CHECK(hipGraphAddKernelNode(&new_node, current->graph, NULL, 0, &k_params));
	} else {
		CHECK(hipGraphAddKernelNode(&new_node, current->graph, {&current->last_node}, 1, &k_params));
	}
	current->last_node = new_node;
}

void Schedule::begin_fixpoint(void){
	current->next = new Subgraph(current->lvl + 1);
	fixpoints.push_back(current->next);
	current = current->next;
}

void Schedule::end_fixpoint(void){
	current->fp_start = fixpoints.back();
	fixpoints.pop_back();
	current->next = new Subgraph(current->lvl - 1);
	current = current->next;
}

hipGraphExec_t Schedule::instantiate(hipStream_t stream){
	hipGraphExec_t result = head->instantiate(launch_kernel, relaunch_fp_kernel);
	head->fill_out_fixpoints(stream, relaunch_fp_kernel);
	return result;
}

void Schedule::print(void){
	head->print(0);
}

Schedule::Schedule(void* launch_kernel, void* relaunch_fp_kernel) :
	launch_kernel(launch_kernel), relaunch_fp_kernel(relaunch_fp_kernel) {
	head = new Subgraph(-1);
	current = head;
}

Schedule::Subgraph::Subgraph(int lvl) : lvl(lvl), next(NULL), last_node(NULL), fp_start(NULL) {
	CHECK(hipGraphCreate(&graph, 0));
}

hipGraphExec_t Schedule::Subgraph::instantiate(void* launch_kernel, void* relaunch_fp_kernel) {
	hipGraphExec_t next_to_launch;
	if(next == NULL) {
		next_to_launch = NULL;
	} else {
		next_to_launch = next->instantiate(launch_kernel, relaunch_fp_kernel);
	}

	// Holds kernel node parameters
	hipKernelNodeParams k_params;
	
	// If we are the end of a fixpoint a conditional relaunch should be added
	if(fp_start != NULL) {
		k_params.func = relaunch_fp_kernel;
		k_params.gridDim = dim3(1, 1, 1);
		k_params.blockDim = dim3(1, 1, 1);
		k_params.sharedMemBytes = 0;
		
		void* n = NULL;
		void* args[3] = {
			(void*)&lvl,
			/*	At this point the start of our fixpoint is 
				not instantiated yet, we revisit this parameter later.*/
			(void*)&n, 
			(void*)&next_to_launch
		};
		k_params.kernelParams = args;
		k_params.extra = NULL;

		hipGraphNode_t new_node;
		if (last_node == NULL){
			CHECK(hipGraphAddKernelNode(&new_node, graph, NULL, 0, &k_params));
		} else {
			CHECK(hipGraphAddKernelNode(&new_node, graph, &last_node, 1, &k_params));
		}
		last_node = new_node;
	} // Otherwise an unconditional launch suffices
	else if (next_to_launch != NULL) {
		// If we have nothing to execute, we directly execute the next step
		if (last_node == NULL){
			graph_exec = next_to_launch;
			return graph_exec;
		} else {
			hipGraphNode_t new_node;
			k_params.func = launch_kernel;
			k_params.gridDim = dim3(1, 1, 1);
			k_params.blockDim = dim3(1, 1, 1);
			k_params.sharedMemBytes = 0;
			void* args[1] = {
				(void*)&next_to_launch
			};
			k_params.kernelParams = args;
			k_params.extra = NULL;
			CHECK(hipGraphAddKernelNode(&new_node, graph, &last_node, 1, &k_params));
			last_node = new_node;
		}
	}

	// The graph is now complete, so we instantiate:
	CHECK(hipGraphInstantiate(&graph_exec, graph, hipGraphInstantiateFlagDeviceLaunch));
	return graph_exec;
}

void Schedule::Subgraph::fill_out_fixpoints(hipStream_t stream, void* relaunch_fp_kernel) {
	// If we have a fixpoint to start, we update our relaunch node.
	if(fp_start != NULL) {
		// Holds kernel node parameters
		hipKernelNodeParams k_params;
		
		k_params.func = relaunch_fp_kernel;
		k_params.gridDim = dim3(1, 1, 1);
		k_params.blockDim = dim3(1, 1, 1);
		k_params.sharedMemBytes = 0;

		hipGraphExec_t next_to_launch;
		if(next == NULL) {
			next_to_launch = NULL;
		} else {
			next_to_launch = next->graph_exec;
		}

		void* args[3] = {
			(void*)&lvl,
			(void*)&fp_start->graph_exec, 
			(void*)&next_to_launch
		};
		k_params.kernelParams = args;
		k_params.extra = NULL;

		CHECK(hipGraphExecKernelNodeSetParams(graph_exec, last_node, &k_params));
	}
	/*	Now that our graph is complete, we upload it.
		We prevent uploading twice if we
		are directly launching next's graph.
	*/
	if(next == NULL || graph_exec != next->graph_exec) {
		CHECK(hipGraphUpload(graph_exec, stream));
	}

	if(next){
		next->fill_out_fixpoints(stream, relaunch_fp_kernel);
	}
}

void Schedule::Subgraph::print(uint graph_num){
	char dot_file[20];
	sprintf(dot_file, "graph%u.dot", graph_num);
	CHECK(hipGraphDebugDotPrint(graph, dot_file, hipGraphDebugDotFlagsKernelNodeParams));
	if(next != NULL) {
		next->print(graph_num + 1);
	}
}